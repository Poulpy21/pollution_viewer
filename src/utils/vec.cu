#include "hip/hip_runtime.h"

#include <cfloat> 
#include "vec.hpp"

__host__ __device__ Vec::Vec() : x(0.0f), y(0.0f), z(0.0f) {}
__host__ __device__ Vec::Vec(const Vec &v) : x(v.x), y(v.y), z(v.z) {}
__host__ __device__ Vec::Vec(float x, float y, float z) : x(x), y(y), z(z) {}
__host__ __device__ Vec::~Vec() {}

__host__ __device__ Vec& Vec::operator= (const Vec &v) {
    Vec V(v);
    return *this;
}


__host__ __device__ void Vec::setValue(float x, float y, float z) {
    this->x = x;
    this->y = y;
    this->z = z;
}

__host__ __device__ Vec & Vec::operator+= (const Vec &a) {
    x += a.x;
    y += a.y;
    z += a.z;
    return *this;
}

__host__ __device__ Vec & Vec::operator-= (const Vec &a) {
    x -= a.x;
    y -= a.y;
    z -= a.z;
    return *this;
}

__host__ __device__ Vec & Vec::operator*= (const Vec &a) {
    x *= a.x;
    y *= a.y;
    z *= a.z;
    return *this;
}

__host__ __device__ Vec & Vec::operator/= (const Vec &a) {
    x /= a.x;
    y /= a.y;
    z /= a.z;
    return *this;
}

__host__ __device__ Vec & Vec::operator^= (const Vec &a) {
    Vec b(*this);
    x = b.y*a.z - b.z*a.y;
    y = b.z*a.x - b.x*a.z;
    z = b.x*a.y - b.y*a.x;
    return *this;
}

__host__ __device__ Vec & Vec::operator+= (float k) {
    x += k;
    y += k;
    z += k;
    return *this;
}

__host__ __device__ Vec & Vec::operator-= (float k) {
    x -= k;
    y -= k;
    z -= k;
    return *this;
}

__host__ __device__ Vec & Vec::operator*= (float k) {
    x *= k;
    y *= k;
    z *= k;
    return *this;
}

__host__ __device__ Vec & Vec::operator/= (float k) {
    x /= k;
    y /= k;
    z /= k;
    return *this;
}

__host__ __device__ Vec operator+ (const Vec &a, const Vec &b) {
    return Vec(a.x+b.x, a.y+b.y, a.z+b.z);
}

__host__ __device__ Vec operator- (const Vec &a, const Vec &b) {
    return Vec(a.x-b.x, a.y-b.y, a.z-b.z);
}

__host__ __device__ Vec operator* (const Vec &a, const Vec &b) {
    return Vec(a.x*b.x, a.y*b.y, a.z*b.z);
}

__host__ __device__ Vec operator/ (const Vec &a, const Vec &b) {
    return Vec(a.x/b.x, a.y/b.y, a.z/b.z);
}

__host__ __device__ Vec operator^ (const Vec &a, const Vec &b) {
    return Vec(
            a.y*b.z - a.z*b.y,
            a.z*b.x - a.x*b.z,
            a.x*b.y - a.y*b.x
            );
}

__host__ __device__ float operator| (const Vec &a, const Vec &b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}

__host__ __device__ Vec operator* (const Vec &a, float k) {
    return Vec(a.x * k, a.y * k, a.z * k);
}

__host__ __device__ Vec operator/ (const Vec &a, float k) {
    return Vec(a.x / k, a.y / k, a.z / k);
}

__host__ __device__ Vec operator* (float k, const Vec &b) {
    return Vec(b.x * k, b.y * k, b.z * k);
}

__host__ __device__ Vec operator/ (float k, const Vec &b) {
    return Vec(b.x / k, b.y / k, b.z / k);
}


__host__ __device__ bool operator!= (const Vec &a, const Vec &b) {
    return !(a == b);
}
__host__ __device__ bool operator== (const Vec &a, const Vec &b) {
    Vec dv = b - a;
    return (abs(dv.x) < FLT_EPSILON && abs(dv.y) < FLT_EPSILON && abs(dv.z) < FLT_EPSILON);
}

__host__ __device__ float Vec::normalize () {
    float norm = this->norm();
    x /= norm;
    y /= norm;
    z /= norm;
    return norm;
}

__host__ __device__ Vec Vec::orthogonalVec () const {
    return Vec(z,z,-x-y);
}

__host__ __device__ Vec Vec::unit () const {
    Vec v(*this);
    v.normalize();
    return v;
}

__host__ __device__ float Vec::squaredNorm () const {
    return x*x + y*y + z*z;
}

__host__ __device__ float Vec::norm () const {
    return sqrt(x*x + y*y + z*z);
}

__host__ std::ostream & operator << (std::ostream &os, Vec &v) {
    os << "(" << v.x << "," << v.y << "," << v.z << ")";
    return os;
}
